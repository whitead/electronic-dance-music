#include "hip/hip_runtime.h"
#include "gaussian_grid_gpu.cuh"

EDM::GaussGrid* EDM::make_gauss_grid_gpu( int dim, 
			   const double* min, 
			   const double* max, 
			   const double* bin_spacing, 
			   const int* b_periodic, 
			   int b_interpolate,
			   const double* sigma) {
  switch(dim) {
  case 1:
    return new DimmedGaussGridGPU<1>(min, max, bin_spacing, b_periodic, b_interpolate, sigma);
  case 2:
    return new DimmedGaussGridGPU<2>(min, max, bin_spacing, b_periodic, b_interpolate, sigma);
  case 3:
    return new DimmedGaussGridGPU<3>(min, max, bin_spacing, b_periodic, b_interpolate, sigma);
  }

  return NULL;
}


EDM::GaussGrid* EDM::read_gauss_grid_gpu( int dim, const std::string& filename, const double* sigma) {
  switch(dim) {
  case 1:
    return new DimmedGaussGridGPU<1>(filename, sigma);
  case 2:
    return new DimmedGaussGridGPU<2>(filename, sigma);
  case 3:
    return new DimmedGaussGridGPU<3>(filename, sigma);
  }
  return NULL;
}
