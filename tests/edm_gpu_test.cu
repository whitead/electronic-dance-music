#include "hip/hip_runtime.h"
#include "grid_gpu.cuh"
#include "edm_bias_gpu.cuh"
#include "gaussian_grid_gpu.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
//These must be declared here.
#define BOOST_TEST_DYN_LINK 
#define BOOST_TEST_MODULE EDM_GPU

#define EPSILON 1e-10
#define QUOTE(name) #name
#define STR(macro) QUOTE(macro)
#define GRID_SRC std::string(STR(TEST_GRID_SRC))

#include <boost/timer/timer.hpp>
#include <boost/chrono.hpp>
#include <boost/test/unit_test.hpp>

#define TIMING_BOUND_edm_cpu_timer_1d 10000

using namespace boost;
using namespace EDM;
using namespace EDM_Kernels;

typedef chrono::duration<double> sec; // seconds, stored with a double




//Many of these test are the same as the serial ones, just to make sure we preserve behavior
BOOST_AUTO_TEST_CASE( grid_gpu_1d_sanity ){
  double min[] = {0};
  double max[] = {10};
  double bin_spacing[] = {1};
  int periodic[] = {0};
  DimmedGridGPU<1> g (min, max, bin_spacing, periodic, 0, 0);
  DimmedGridGPU<1>* d_g;
  BOOST_REQUIRE_EQUAL(g.grid_number_[0], 11);
  BOOST_REQUIRE_EQUAL(g.grid_size_, 11);
  gpuErrchk(hipMalloc((void**) &d_g, sizeof(DimmedGridGPU<1>)));

  size_t array[] = {5};
  size_t temp[1];
  g.one2multi(g.multi2one(array), temp);
  BOOST_REQUIRE_EQUAL(array[0], temp[0]);

  for(int i = 0; i < 11; i++){
    g.grid_[i] = i;
  }
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(d_g, &g, sizeof(DimmedGridGPU<1>), hipMemcpyHostToDevice));
  gpuErrchk(hipDeviceSynchronize());
  double x[] = {3.5};
  BOOST_REQUIRE(g.in_grid(x));
  size_t index[1];
  g.get_index(x, index);
  BOOST_REQUIRE(index[0] - 3 < 0.000001);

  double* d_x;
  gpuErrchk(hipMalloc(&d_x, sizeof(double)));
  gpuErrchk(hipMemcpy(d_x, x, sizeof(double), hipMemcpyHostToDevice));
  double target[1] = {0.0};
  double* d_target;
  gpuErrchk(hipMalloc((void**) &d_target, sizeof(double)));
  gpuErrchk(hipMemcpy(d_target, target, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipDeviceSynchronize());
  get_value_kernel<1><<<1,1>>>(d_x, d_target, d_g);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(target, d_target, sizeof(double), hipMemcpyDeviceToHost));
  BOOST_REQUIRE(pow(target[0] -3, 2) < 0.000001);

  //try to break it
  x[0] = 0;
  
  gpuErrchk(hipMemcpy(d_x, x, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipDeviceSynchronize());
  get_value_kernel<1><<<1,1>>>(d_x, d_target, d_g);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(target, d_target, sizeof(double), hipMemcpyDeviceToHost));
  BOOST_REQUIRE(g.get_value(x) - target[0] < EPSILON);//require same behavior on host/dev

  x[0] = 10;
  
  gpuErrchk(hipMemcpy(d_x, x, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipDeviceSynchronize());
  get_value_kernel<1><<<1,1>>>(d_x, d_target, d_g);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(target, d_target, sizeof(double), hipMemcpyDeviceToHost));
  BOOST_REQUIRE(g.get_value(x) - target[0] < EPSILON);//require same behavior on host/dev

  gpuErrchk(hipFree(d_g));
  gpuErrchk(hipFree(d_target));
  gpuErrchk(hipFree(d_x));
}

BOOST_AUTO_TEST_CASE( grid_gpu_3d_sanity )
{//must now refactor this test to use kernels.
  double min[] = {-2, -5, -3};
  double max[] = {125, 63, 78};
  double bin_spacing[] = {1.27, 1.36, 0.643};
  int periodic[] = {0, 1, 1};
  DimmedGridGPU<3> g (min, max, bin_spacing, periodic, 0, 0);
  DimmedGridGPU<3>* d_g;
  BOOST_REQUIRE_EQUAL(g.grid_number_[0], 101);
  BOOST_REQUIRE_EQUAL(g.grid_number_[1], 50);
  BOOST_REQUIRE_EQUAL(g.grid_number_[2], 126);
  gpuErrchk(hipMalloc((void**) &d_g, sizeof(DimmedGridGPU<3>)));
  gpuErrchk(hipMemcpy(d_g, &g, sizeof(DimmedGridGPU<3>), hipMemcpyHostToDevice));

  size_t array[3];
  size_t temp[3];
  size_t* d_array;
  size_t* d_temp;
  gpuErrchk(hipMalloc((void**)&d_array, 3*sizeof(size_t)));
  gpuErrchk(hipMalloc((void**)&d_temp, 3*sizeof(size_t)));

  for(int i = 0; i < g.grid_number_[0]; i++) {
    array[0] = i;
    for(int j = 0; j < g.grid_number_[1]; j++) {
      array[1] = j;
      for(int k = 0; k < g.grid_number_[2]; k++) {
	array[2] = k;
	  /* This passes but it's slow, so leaving out for now
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(d_array, array, 3*sizeof(size_t), hipMemcpyHostToDevice));

	//g.one2multi(g.multi2one(array), temp);
	multi2one_kernel<3><<<1,1>>>(d_g, d_array, d_temp);
	gpuErrchk(hipDeviceSynchronize());
	gpuErrchk(hipMemcpy(array, d_array, 3*sizeof(size_t), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(temp, d_temp, 3*sizeof(size_t), hipMemcpyDeviceToHost));
//	gpuErrchk(hipDeviceSynchronize());
	BOOST_REQUIRE_EQUAL(array[0], temp[0]);
	BOOST_REQUIRE_EQUAL(array[1], temp[1]);
	BOOST_REQUIRE_EQUAL(array[2], temp[2]);
*/
	g.grid_[g.multi2one(array)] = g.multi2one(array);
      }
    }
  }
  
  double point[3];
  gpuErrchk(hipDeviceSynchronize());
  for(int i = 0; i < g.grid_number_[0]; i++) {
    point[0] = i * g.dx_[0] + g.min_[0] + EPSILON;
    array[0] = i;
    for(int j = 0; j < g.grid_number_[1]; j++) {
      point[1] = j * g.dx_[1] + g.min_[1] + EPSILON;
      array[1] = j;
      for(int k = 0; k < g.grid_number_[2]; k++) {
	point[2] = k * g.dx_[2] + g.min_[2] + EPSILON;
	array[2] = k;
	BOOST_REQUIRE(pow(g.do_get_value(point) - g.grid_[g.multi2one(array)],2) < 0.0000001);
      }
    }
  }
  gpuErrchk(hipFree(d_g));
  gpuErrchk(hipFree(d_temp));
  gpuErrchk(hipFree(d_array));
}

BOOST_AUTO_TEST_CASE( grid_gpu_1d_read ) {
  DimmedGridGPU<1> g(GRID_SRC + "/1.grid");
  BOOST_REQUIRE_EQUAL(g.min_[0], 0);
  BOOST_REQUIRE_EQUAL(g.max_[0], 2.5 + g.dx_[0]);
  BOOST_REQUIRE_EQUAL(g.grid_number_[0], 101);
}

BOOST_AUTO_TEST_CASE( grid_gpu_3d_read ) {
  DimmedGridGPU<3> g(GRID_SRC + "/3.grid");//derivatives is true here
  DimmedGridGPU<3>* d_g;
  gpuErrchk(hipMalloc((void**) &d_g, sizeof(DimmedGridGPU<3>)));
  gpuErrchk(hipMemcpy(d_g, &g, sizeof(DimmedGridGPU<3>), hipMemcpyHostToDevice));

  BOOST_REQUIRE_EQUAL(g.min_[2], 0);
  BOOST_REQUIRE_EQUAL(g.max_[2], 2.5 + g.dx_[2]);
  BOOST_REQUIRE_EQUAL(g.grid_number_[2], 11);
  double temp[] = {0.75, 0, 1.00};
  double* d_temp;
  double* d_target;
  double target[1] = {0.0};
  
  gpuErrchk(hipMalloc((void**)&d_temp, 3*sizeof(double)));
  gpuErrchk(hipMalloc((void**) &d_target, sizeof(double)));
  gpuErrchk(hipMemcpy(d_temp, temp, 3*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_target, target, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipDeviceSynchronize());
  
  get_value_kernel<3><<<1,1>>>(d_temp, d_target, d_g);
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(target, d_target, sizeof(double), hipMemcpyDeviceToHost));

  BOOST_REQUIRE(pow(target[0] - 1.260095, 2) < EPSILON);

  gpuErrchk(hipFree(d_g));
  gpuErrchk(hipFree(d_temp));
  gpuErrchk(hipFree(d_target));
}

BOOST_AUTO_TEST_CASE( gpu_derivative_direction ) {
  DimmedGridGPU<3> g(GRID_SRC + "/3.grid", 1);
  DimmedGridGPU<3>* d_g;
  gpuErrchk(hipMalloc((void**) &d_g, sizeof(DimmedGridGPU<3>)));
  gpuErrchk(hipMemcpy(d_g, &g, sizeof(DimmedGridGPU<3>), hipMemcpyHostToDevice));
  double temp[] = {0.75, 0, 1.00};
  double temp2[] = {0.76, 0, 1.00};
  double target[]={0.0};
  double target2[]={0.0};
  double* d_temp;
  double* d_temp2;
  double* d_target;
  double* d_target2;
  gpuErrchk(hipMalloc((void**)&d_temp, 3*sizeof(double)));
  gpuErrchk(hipMalloc((void**) &d_target, sizeof(double)));
  gpuErrchk(hipMalloc((void**)&d_temp2, 3*sizeof(double)));
  gpuErrchk(hipMalloc((void**) &d_target2, sizeof(double)));
  
  gpuErrchk(hipMemcpy(d_temp, temp, 3*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_target, target, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_temp2, temp2, 3*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_target2, target2, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipDeviceSynchronize());
  
  get_value_kernel<3><<<1,1>>>(d_temp, d_target, d_g);
  get_value_kernel<3><<<1,1>>>(d_temp2, d_target2, d_g);
  
  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(temp, d_temp, 3*sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(target, d_target, sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(temp2, d_temp2, 3*sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(target2, d_target2, sizeof(double), hipMemcpyDeviceToHost));

  BOOST_REQUIRE(target2[0] > target[0]);
  
  temp2[0] = 0.75;
  temp2[2] = 0.99;
  target[0] = 0.0;
  target2[0] = 0.0;

  gpuErrchk(hipMemcpy(d_temp, temp, 3*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_target, target, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_temp2, temp2, 3*sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_target2, target2, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipDeviceSynchronize());

  get_value_kernel<3><<<1,1>>>(d_temp, d_target, d_g);
  get_value_kernel<3><<<1,1>>>(d_temp2, d_target2, d_g);

  gpuErrchk(hipDeviceSynchronize());
  gpuErrchk(hipMemcpy(temp, d_temp, 3*sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(target, d_target, sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(temp2, d_temp2, 3*sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(target2, d_target2, sizeof(double), hipMemcpyDeviceToHost));
  
  BOOST_REQUIRE(target2[0] < target[0]);
  gpuErrchk(hipFree(d_temp));
  gpuErrchk(hipFree(d_target));
  gpuErrchk(hipFree(d_temp2));
  gpuErrchk(hipFree(d_target2));
  gpuErrchk(hipFree(d_g));
}

BOOST_AUTO_TEST_CASE( grid_gpu_read_write_consistency ) {
  size_t i, j;
  std::string input;
  std::string output;
  for(i = 1; i <= 3; i++) {
    std::stringstream filename;
    filename << i << ".grid";
    input = GRID_SRC + "/" + filename.str();
    output = filename.str() + ".test";
    Grid* g;
    switch(i) {
    case 1:
      g = new DimmedGridGPU<1>(input);
      break;
    case 2:
      g = new DimmedGridGPU<2>(input);
      break;
    case 3:
      g = new DimmedGridGPU<3>(input);
      break;
    }
    g->write(output);
    //grab the grid for comparison
    size_t ref_length = g->get_grid_size();
    double ref_grid[ref_length];
    for(j = 0; j < ref_length; j++)
      ref_grid[j] = g->get_grid()[j];
    //re-read
    gpuErrchk(hipDeviceSynchronize());
    g->read(output);
    //now compare
    BOOST_REQUIRE_EQUAL(g->get_grid_size(), ref_length);

    for(j = 0; j < ref_length; j++)
      BOOST_REQUIRE(pow(ref_grid[j] - g->get_grid()[j], 2) < EPSILON);

  }
}

BOOST_AUTO_TEST_CASE( gpu_interpolation_1d ) {
  
  double min[] = {0};
  double max[] = {10};
  double bin_spacing[] = {1};
  int periodic[] = {0};
  DimmedGridGPU<1> g (min, max, bin_spacing, periodic, 1, 1);
  
  for(int i = 0; i < 11; i++) {
    g.grid_[i] = log(i);
    g.grid_deriv_[i] = 1. / i;
  }

  DimmedGridGPU<1>* d_g;
  gpuErrchk(hipMalloc((void**) &d_g, sizeof(DimmedGridGPU<3>)));
  gpuErrchk(hipMemcpy(d_g, &g, sizeof(DimmedGridGPU<1>), hipMemcpyHostToDevice));

  double array[] = {5.3};
  double der[1];
  double fhat[1] = {0.0};//g.get_value_deriv(array,der);
  double* d_array;
  double* d_der;
  double* d_fhat;
  gpuErrchk(hipMalloc((void**)&d_array, sizeof(double)));
  gpuErrchk(hipMalloc((void**)&d_der, sizeof(double)));
  gpuErrchk(hipMalloc((void**)&d_fhat, sizeof(double)));

  gpuErrchk(hipMemcpy(d_array, array, sizeof(double), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(d_der, der, sizeof(double), hipMemcpyHostToDevice ));
  gpuErrchk(hipMemcpy(d_fhat, fhat, sizeof(double), hipMemcpyHostToDevice ));

  get_value_deriv_kernel<1><<<1,1>>>(d_array, d_der, d_fhat, d_g);

  gpuErrchk(hipMemcpy(array, d_array, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(der, d_der, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(fhat, d_fhat, sizeof(double), hipMemcpyDeviceToHost ));

  
  //make sure it's at least in the ballpark
  BOOST_REQUIRE(fhat[0] > log(5) && fhat[0] < log(6));
  BOOST_REQUIRE(der[0] < 1. / 5 && der[0] > 1. / 6.);

  //Make sure it's reasonably accurate
  BOOST_REQUIRE(pow(fhat[0] - log(5.3), 2) < 0.1);
  BOOST_REQUIRE(pow(der[0]- 1. / 5.3, 2) < 0.1);

  //try edge cases
  array[0] = 5.0;
  gpuErrchk(hipMemcpy(d_array, array, sizeof(double), hipMemcpyHostToDevice ));
  get_value_deriv_kernel<1><<<1,1>>>(d_array, d_der, d_fhat, d_g);

  gpuErrchk(hipMemcpy(array, d_array, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(der, d_der, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(fhat, d_fhat, sizeof(double), hipMemcpyDeviceToHost ));

  array[0] = 5.5;
  gpuErrchk(hipMemcpy(d_array, array, sizeof(double), hipMemcpyHostToDevice ));
  get_value_deriv_kernel<1><<<1,1>>>(d_array, d_der, d_fhat, d_g);

  gpuErrchk(hipMemcpy(array, d_array, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(der, d_der, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(fhat, d_fhat, sizeof(double), hipMemcpyDeviceToHost ));

  array[0] = 0.0;
  gpuErrchk(hipMemcpy(d_array, array, sizeof(double), hipMemcpyHostToDevice ));
  get_value_deriv_kernel<1><<<1,1>>>(d_array, d_der, d_fhat, d_g);

  gpuErrchk(hipMemcpy(array, d_array, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(der, d_der, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(fhat, d_fhat, sizeof(double), hipMemcpyDeviceToHost ));

  array[0] = 10.0;
  gpuErrchk(hipMemcpy(d_array, array, sizeof(double), hipMemcpyHostToDevice ));
  get_value_deriv_kernel<1><<<1,1>>>(d_array, d_der, d_fhat, d_g);

  gpuErrchk(hipMemcpy(array, d_array, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(der, d_der, sizeof(double), hipMemcpyDeviceToHost ));
  gpuErrchk(hipMemcpy(fhat, d_fhat, sizeof(double), hipMemcpyDeviceToHost ));


}

BOOST_AUTO_TEST_CASE( gpu_interp_1d_periodic ) {
  double min[] = {-M_PI};
  double max[] = {M_PI};
  double bin_spacing[] = {M_PI / 100};
  int periodic[] = {1};
  DimmedGridGPU<1> g (min, max, bin_spacing, periodic, 1, 1);

  for(int i = 0; i < g.grid_number_[0]; i++) {
    g.grid_[i] = sin(g.min_[0] + i * g.dx_[0]);
    g.grid_deriv_[i] = cos(g.min_[0] + i * g.dx_[0]);
  }
  DimmedGridGPU<1>* d_g;
  gpuErrchk(hipMalloc((void**) &d_g, sizeof(DimmedGridGPU<1>)));
  gpuErrchk(hipMemcpy(d_g, &g, sizeof(DimmedGridGPU<1>), hipMemcpyHostToDevice));


  double array[] = {M_PI / 4};
  double der[1];
  double fhat[1] = {0.0};//g.get_value_deriv(array,der);

  double* d_array;
  double* d_der;
  double* d_fhat;
  gpuErrchk(hipMalloc((void**)&d_array, sizeof(double)));
  gpuErrchk(hipMalloc((void**)&d_der, sizeof(double)));
  gpuErrchk(hipMalloc((void**)&d_fhat, sizeof(double)));

  gpuErrchk(hipMemcpy((void**)d_array, array, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy((void**)d_fhat, fhat, sizeof(double), hipMemcpyHostToDevice));

  get_value_deriv_kernel<1><<<1,1>>>(d_array, d_der, d_fhat, d_g);
  
  gpuErrchk(hipMemcpy(fhat, d_fhat, sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(der, d_der, sizeof(double), hipMemcpyDeviceToHost));
  
  //Make sure it's reasonably accurate
  BOOST_REQUIRE(pow(fhat[0] - sin(array[0]), 2) < 0.1);
  BOOST_REQUIRE(pow(der[0] - cos(array[0]), 2) < 0.1);

  //test periodic
  array[0] = 5 * M_PI / 4;
  gpuErrchk(hipMemcpy((void**)d_array, array, sizeof(double), hipMemcpyHostToDevice));
  get_value_deriv_kernel<1><<<1,1>>>(d_array, d_der, d_fhat, d_g);
  gpuErrchk(hipMemcpy(fhat, d_fhat, sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(der, d_der, sizeof(double), hipMemcpyDeviceToHost));
  
  //fhat = g.get_value_deriv(array,der);

  g.write("grid.test");
  
  BOOST_REQUIRE(pow(fhat[0] - sin(array[0]), 2) < 0.1);
  BOOST_REQUIRE(pow(der[0] - cos(array[0]), 2) < 0.1);

}




//This test will simply run several thousand timesteps and time how long it takes.
BOOST_AUTO_TEST_CASE( edm_cpu_timer_1d ){
  
  double min[] = {-10};
  double max[] = {10};
  double sigma[] = {1};
  double bin_spacing[] = {1};
  int periodic[] = {1};
  double x[1] = {0};
  unsigned int n_hills = 5000;
  DimmedGaussGrid<1> g (min, max, bin_spacing, periodic, 0, sigma);
  //now just do a generic loop, adding 10k gaussians, and time it
  boost::timer::auto_cpu_timer t;
  for( unsigned int i = 0; i < n_hills; i++){
    int rand_num = rand() % 20 - 10;
    x[0] = rand_num;
    g.add_value(x,1);
  }
  t.stop();
  sec seconds = chrono::nanoseconds(t.elapsed().user);
  
  BOOST_REQUIRE(seconds.count() < TIMING_BOUND_edm_cpu_timer_1d);
}

//BOOST_AUTO_TEST_SUITE_END()

