#include "hip/hip_runtime.h"
#include "grid.cuh"
#include "edm_bias.cuh"
#include "gaussian_grid.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
//These must be declared here.
#define BOOST_TEST_DYN_LINK 
#define BOOST_TEST_MODULE EDM_GPU

#include <boost/timer/timer.hpp>
#include <boost/chrono.hpp>
#include <boost/test/unit_test.hpp>

#define TIMING_BOUND_edm_cpu_timer_1d 10000

using namespace boost;
using namespace EDM;

typedef chrono::duration<double> sec; // seconds, stored with a double

BOOST_AUTO_TEST_SUITE( edm_gpu )

//This test will simply run several thousand timesteps and time how long it takes.
BOOST_AUTO_TEST_CASE( edm_cpu_timer_1d ){
  
  double min[] = {-10};
  double max[] = {10};
  double sigma[] = {1};
  double bin_spacing[] = {1};
  int periodic[] = {1};
  double x[1] = {0};
  unsigned int n_hills = 5000;
  DimmedGaussGrid<1> g (min, max, bin_spacing, periodic, 0, sigma);
  //now just do a generic loop, adding 10k gaussians, and time it
  boost::timer::auto_cpu_timer t;
  for( unsigned int i = 0; i < n_hills; i++){
    int rand_num = rand() % 20 - 10;
    x[0] = rand_num;
    g.add_value(x,1);
  }
  t.stop();
  sec seconds = chrono::nanoseconds(t.elapsed().user);
  
  BOOST_REQUIRE(seconds.count() < TIMING_BOUND_edm_cpu_timer_1d);
}

BOOST_AUTO_TEST_SUITE_END()

